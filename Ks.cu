#include "hip/hip_runtime.h"
#include "Ks.cuh"
#include <iostream>
#include<stdio.h>
#include<algorithm>
using namespace std;
unsigned int iDivUp(unsigned int a, unsigned int b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);	
}
class CThreadScaler
{
private:
	int Dg;
	int Db;
public:
	CThreadScaler(int NumThreads)
	{
		Db = min( 512, NumThreads);
		if(Db > 0)
		{
			Dg = iDivUp(NumThreads, Db);
		}else
		{
			Dg = 0;
		}
	}
	int Grids()
	{
		return Dg;
	}
	int Blocks()
	{
		return Db;
	}
};
static __inline__ __device__ int CudaGetTargetID()
{
	return blockDim.x * blockIdx.x + threadIdx.x;
}
__global__ void knapsackKernel(int *V,int *W,int *M,int Capacity,int i)
{
	int w = CudaGetTargetID();
	if (w > Capacity)
		return;
	if (i%2!=0)
	{
		if (W[i] <= w)
			M[Capacity+w] = max(V[i] + M[w-W[i]],M[w] );
        else
             M[Capacity+w] = M[w];
    }
	else
	{
		if (W[i] <= w)
			M[w] = max(V[i] + M[Capacity + (w-W[i])],M[Capacity + w]);
        else
            M[w] = M[Capacity+w];
	}
	
	__syncthreads();
}

void CudaFunctionCall(int *Value,int *Weight,int *Matrix,int NumofItems,int Capacity)
{
	CudaSafeCall(hipSetDevice(0));
	CThreadScaler TS(Capacity+1);
	int i=0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	while (i <NumofItems) 
	{
		knapsackKernel<<<TS.Grids(), TS.Blocks()>>>(Value, Weight, Matrix,Capacity,i);
		CudaSafeCall(hipDeviceSynchronize());
		i++;
	}
	hipEventRecord(stop);
	CudaSafeCall(hipGetLastError());
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Parallel Time in ms:%f\n",milliseconds);
	
	
}
